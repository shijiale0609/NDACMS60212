#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <assert.h>

//#include <new>
//#include <cstddef>
//#include <fstream>
//#include "pointer_2d_matrix.h"
#include "cuda_wrapper.h"

// Thread block size
//#define BLOCK_SIZE  16  // number of threads in a direction of the block
//#define M_WIDTH     512 // number of columns
//#define M_HEIGHT    512 // number of rows

#define TILE_W      14
#define TILE_H      14
#define R           1                   // filter radius
#define D           (R*2+1)             // filter diameter
#define S           (D*D)               // filter size
#define BLOCK_W     (TILE_W+(2*R))
#define BLOCK_H     (TILE_H+(2*R))


// I take the following links as my reference for cuda_kernel_wrapper
// https://stackoverflow.com/questions/39336574/box-filter-with-cuda-c
// https://www.nvidia.com/content/nvision2008/tech_presentations/Game_Developer_Track/NVISION08-Image_Processing_and_Video_with_CUDA.pdf
__global__ void d_filter(float *g_idata, float *g_odata, unsigned int width, unsigned int height)
{

  __shared__ int smem[BLOCK_W*BLOCK_H*BLOCK_H];

    int x = blockIdx.x*TILE_W + threadIdx.x - R;
    int y = blockIdx.y*TILE_H + threadIdx.y - R;
    // clamp to edge of image
    x = max(0, x);
    x = min(x, width-1);
    y = max(y, 0);
    y = min(y, height-1);
    unsigned int index = y*width + x;
    unsigned int bindex = threadIdx.y*blockDim.y+threadIdx.x;
    // each thread copies its pixel of the block to shared memory
    smem[bindex] = g_idata[index];
    __syncthreads();
    // only threads inside the apron will write results
    if ((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W-R)) && (threadIdx.y >= R) && (threadIdx.y < (BLOCK_H-R))) {
        float sum = 0;
        for(int dy=-R; dy<=R; dy++) {
            for(int dx=-R; dx<=R; dx++) {
                float i = smem[bindex + (dy*blockDim.x) + dx];
                sum += i;
            }
        }
        g_odata[index] = sum / S;
    }

}

int cuda_kernel_wrapper(float *data, int image_Width, int image_Height){

    float   *d_idata, *d_odata;
    unsigned int w = image_Width, h = image_Height , channels= 1;

    //printf("initial results:\n");
    //for (int i = 1000; i < 1016; i++){
    //  for (int j = 1000; j < 1016; j++) printf("%f ", data[i*w+j]);
    //  printf("\n");}

    unsigned int numElements = w*h*channels;
    size_t datasize = numElements * sizeof(float);
    hipError_t err = hipSuccess;    

    // Allocate the Device Memory
    printf("Allocate Devicememory for data\n");

    err = hipMalloc((void **)&d_idata, datasize);
    if ( err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for idata (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_odata, datasize);
    if ( err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for odata (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy to device
    printf("Copy idata from the host memory to the CUDA device\n");
    err =hipMemcpy(d_idata, data, datasize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy idata from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch Kernel
    dim3 threadsPerBlock(BLOCK_W, BLOCK_H);
    int GRID_W = w/TILE_W+1;
    int GRID_H = h/TILE_H+1;
    dim3 blocksPerGrid(GRID_W, GRID_H);
    //dim3 blocksPerGrid((w+threadsPerBlock.x-1)/threadsPerBlock.x, (h+threadsPerBlock.y-1)/threadsPerBlock.y);
    printf("CUDA kernel launch with %d,%d blocks of %d,%d threads\n", blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // kernel call
    d_filter<<<blocksPerGrid, threadsPerBlock>>>(d_idata, d_odata, w,h);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapseTime;
    hipEventElapsedTime(&elapseTime, start, stop); 
    
    printf("Time to run the kernel: %f ms\n", elapseTime);

    // Copy data from device to host
    printf("Copy odata from the CUDA device to the host memory\n");
    hipMemcpy(data, d_odata, datasize, hipMemcpyDeviceToHost);

    // Free Device memory
    printf("Free Device memory\n");
    hipFree(d_idata);
    hipFree(d_odata);

    //printf("final results:\n");
    //for (int i = 1000; i < 1016; i++){
    //  for (int j = 1000; j < 1016; j++) printf("%f ", data[i*w+j]);
    //  printf("\n");}


    //Free Host memory
    //printf("Free Host memory\n");
    //free(data);

    printf("\nDone\n");

    return 0;
}
